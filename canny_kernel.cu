#include "hip/hip_runtime.h"
#include "canny_kernel.cuh"

__global__ 
void gaussian(int *img, int *origin, int rows, int cols) {
    int gaussianMask[5][5] = {
                                {1, 4, 7, 4, 1},
                                {4, 16, 26, 16, 4},
                                {7, 26, 41, 26, 7},
                                {4, 16, 26, 16, 4},
                                {1, 4, 7, 4, 1}
                               };
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < rows * cols; i += stride) {
        int curRow = i / cols;
        int curCol = i % cols;
        int newPixel = 0;
        for(int rowOffset = -2; rowOffset <= 2; rowOffset ++) {
          for(int colOffset = -2; colOffset <= 2; colOffset ++) {
              int neighbourRow = curRow + rowOffset;
              int neighbourCol = curCol + colOffset;
              if(neighbourRow < 0 || neighbourRow >= rows || neighbourCol < 0 || neighbourCol >= cols) {
                continue;
              }
              int neighbourIndex = neighbourRow * cols + neighbourCol;
              newPixel += origin[neighbourIndex] * gaussianMask[2 + rowOffset][2 + colOffset];
          }
        }
        img[i] = newPixel / 273;
    }
    
}

__global__
void gradient(int *strength, int *direction, int *origin, int rows, int cols) {
    int gxMask[3][3] = {
                         {-1, 0, 1}, 
                         {-2, 0, 2}, 
                         {-1, 0, 1}
                        };
    int gyMask[3][3] = {
                         {-1, -2, -1}, 
                         {0, 0, 0}, 
                         {1, 2, 1}
                        };
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < rows * cols; i += stride) {
      int curRow = i / cols;
      int curCol = i % cols;
      int gx = 0;
      int gy = 0;
      for(int rowOffset = -1; rowOffset <= 1; rowOffset ++) {
        for(int colOffset = -1; colOffset <= 1; colOffset ++) {
          int neighbourRow = curRow + rowOffset;
          int neighbourCol = curCol + colOffset;
          if(neighbourRow < 0 || neighbourRow >= rows || neighbourCol < 0 || neighbourCol >= cols) {
            continue;
          }
          gx += origin[neighbourRow * cols + neighbourCol] * gxMask[rowOffset + 1][colOffset + 1];
          gy += origin[neighbourRow * cols + neighbourCol] * gyMask[rowOffset + 1][colOffset + 1]; 
        }
      }
      strength[i] = sqrtf(gx * gx + gy * gy);
      double angle = (atan2(float(gx), float(gy)) / M_PI) * 180.0;
      if ( ( (angle < 22.5) && (angle > -22.5) ) || (angle > 157.5) || (angle < -157.5))
				direction[i] = 0;
			else if ( ( (angle > 22.5) && (angle < 67.5) ) || ( (angle < -112.5) && (angle > -157.5) ) )
				direction[i] = 45;
			else if ( ( (angle > 67.5) && (angle < 112.5) ) || ( (angle < -67.5) && (angle > -112.5) ) )
				direction[i] = 90;
			else if ( ( (angle > 112.5) && (angle < 157.5) ) || ( (angle < -22.5) && (angle > -67.5) ) )
				direction[i] = 135;
    }
}

__device__
void findEdge(int *strength, int *direction, int *edge, int rows, int cols, 
              int rowShift, int colShift, int i, int dir, int lowerThreshold, int* visited) {
	bool edgeEnd = false;
  int newRow = (i / cols) + rowShift;
	int newCol = (i % cols) + colShift;
  if(newRow < 0 || newRow >= rows) {
      edgeEnd = true;
  }
  if(newCol < 0 || newCol >= cols) {
      edgeEnd = true;
  }
  int idx = newRow * cols + newCol;
  if(visited[idx] == 1) {
    return;
  } else {
    visited[idx] = 1;
  }
  while((direction[idx] == dir) && !edgeEnd &&
      (strength[idx] > lowerThreshold)) {
          edge[idx] = 255;
          newRow = newRow + rowShift;
          newCol = newCol + colShift;
          idx = newRow * cols + newCol;
          if(newRow < 0 || newRow >= rows) {
              break;
          }
          if(newCol < 0 || newRow >= cols) {
              break;
          }
          if(visited[idx] == 1) {
            return;
          } else {
            visited[idx] = 1;
          }
  }
}

__global__
void traceEdge(int *strength, int *direction, int *edge, 
                int rows, int cols, int upperThreshold, int lowerThreshold, int *visited) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;


    for(int i = index; i < rows * cols; i += stride) {
      if(strength[i] > upperThreshold) {
          switch(direction[i]) {
              case 0:
                  findEdge(strength, direction, edge, rows, cols, 0, 1, i, 0, lowerThreshold, visited);
                  break;
              case 45:
                  findEdge(strength, direction, edge, rows, cols, 1, 1, i, 45, lowerThreshold, visited);
                  break;
              case 90:
                  findEdge(strength, direction, edge, rows, cols, 1, 0, i, 90, lowerThreshold, visited);
                  break;
              case 135:
                  findEdge(strength, direction, edge, rows, cols, 1, -1, i, 135, lowerThreshold, visited);
                  break;
              default :
                  edge[i] = 0;     
          }
          edge[i] = edge[i] == 255 ? 255 : 0;
      } else {
          edge[i] = 0;     
      }
    }
}

void canny(int *imageLine, int rows, int cols){

  int *img = NULL, *origin = NULL;

  hipMallocManaged(&img, rows*cols*sizeof(int));
  hipMallocManaged(&origin, rows * cols * sizeof(int));
  hipMemcpy(origin,imageLine,rows * cols * sizeof(int),hipMemcpyHostToDevice);
 
  int numBlocks = (rows * cols + BLOCK_SIZE - 1) / BLOCK_SIZE;

  //gaussian_filter  
  gaussian<<<numBlocks, BLOCK_SIZE>>>(img, origin, rows, cols);
  hipDeviceSynchronize();


  int *strength = NULL, *direction = NULL;
  hipMallocManaged(&strength, rows*cols*sizeof(int));
  hipMallocManaged(&direction, rows * cols * sizeof(int));
  if(strength == NULL || direction == NULL) {
    cout << "GPU Malloc Failed." << endl;
    return;
  }
  gradient<<<numBlocks, BLOCK_SIZE>>>(strength, direction, img, rows, cols);
  hipDeviceSynchronize();
  int *visited;
  hipMallocManaged(&visited, rows*cols*sizeof(int));
  traceEdge<<<numBlocks, BLOCK_SIZE>>>(strength, direction, img, rows, cols, 100, 35, visited);
  hipDeviceSynchronize();

  hipMemcpy(imageLine, img, rows * cols * sizeof(int), hipMemcpyDeviceToHost);
  hipFree(img);
  hipFree(origin);
  hipFree(strength);
  hipFree(direction);

}